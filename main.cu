#include "hip/hip_runtime.h"

int main(){

    // Define variables
    int *U, *V;
    int *dev_U, *dev_U;
    int *partialSum;

    // print info about the system
    int count;
    hipGetDeviceCount( &count );
    printf("there are %d device(s)\n", count);
    for (int i=0; i<count; ++i) {
        hipGetDeviceProperties( &prop, i );
        printf("name is %s\n", prop.name);
        printf("warp size is %d\n", prop.warpSize);
        printf("maxThreadsPerBlock is %d\n", prop.maxThreadsPerBlock);
        printf("maxThreadsDim is (%d, %d, %d)\n", prop.maxThreadsDim[0],
               prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("maxGridSize is (%d, %d, %d)\n", prop.maxGridSize[0],
               prop.maxGridSize[1], prop.maxGridSize[2]);
    }


    U = (float *) malloc(N * sizeof(float));
    V = (float *) malloc(N * sizeof(float));

    int threadsPerBlock = 256;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    // allocate memory on the GPU
    HANDLE_ERROR( hipMalloc( (void **) &dev_U, N*sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void **) &dev_V, N*sizeof(int) ) );

    partialSum = (float *) malloc(numBlocks* sizeof(float));

    for (int i=0; i<N; ++i) {
        U[i] = (float) (i+1);
        V[i] = 1.0 / U[i];
    }

    HandleError( hipMemcpy( dev_U, U, N*sizeof(float), hipMemcpyHostToDevice) );
    HandleError( hipMemcpy( dev_V, V, N*sizeof(float), hipMemcpyHostToDevice) );
    dotp<<<numBlocks, threadsPerBlock, blockSize* sizeof(float)>>>( dev_U, dev_V, dev_partialSum, N );
    hipDeviceSynchronize(); // wait for GPU threads to complete; again, not necessary but good pratice
    HandleError( hipMemcpy( partialSum, dev_partialSum, numBlocks*sizeof(float), hipMemcpyDeviceToHost) );

    // finish up on the CPU side
    float gpuResult= 0.0;
    for (int i=0; i<numBlocks; ++i) gpuResult= gpuResult+ partialSum[i];


    hipFree( dev_U );
    hipFree( dev_V );
    free(U);
    free(V);

    printf("%s\n", gpuResult);
    printf("%s\n", partialSum);
    printf("%s\n", partialSumarray);
    print("DONE");
    // END OF PROGRAM
}

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

__global__ void dotp( float *U, float *V, float *partialSum, int N ) {
    //__shared__ float localCache[BLOCK_SIZE];
    extern __shared__ float localCache[];
    int tidx= threadIdx.x; // my position in my threadblock
    localCache[tidx] = U[tidx] * V[tidx];
    __syncthreads();

    // now, we need to add up the values in localCache[]
    if (threadIdx.x== 0) {
        float temp = 0.0;
        for (int i=0; i<blockDim.x; ++i) temp = temp + localCache[i];localCache[0] = temp;
    }
    // now put the result (this thread block's partial sum) in the partialSumarray
    cacheIndex= threadIdx.x;
    int i= blockDim.x/2;
    while (i> 0) {
        if (cacheIndex< i) localCache[cacheIndex] = localCache[cacheIndex] + localCache[cacheIndex+ i];
        __syncthreads();
        i= i/ 2;
    }

    if (cacheIndex== 0) partialSum[blockIdx.x] = localCache[cacheIdx];
}
